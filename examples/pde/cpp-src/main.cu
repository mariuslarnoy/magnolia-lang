#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include <iostream>
#include "gen/examples/pde/mg-src/pde-cpp.cuh"
#include "base.cuh"

//typedef array_ops::Shape Shape;

static const double s_dt = 0.00082212448155679772495;
static const double s_nu = 1.0;
static const double s_dx = 1.0;

/*
template<>
Array forall_ops<Array,Axis,Float,Index,Nat,Offset,PDEProgram::_snippet_ix>::forall_ix_snippet_cuda(const Array &u, const Array &v,
const Array &u0, const Array &u1, const Array &u2, const Float &c0,
const Float &c1, const Float &c2, const Float &c3, const Float &c4) {

    
  }
*/
template<class _PDEProgram>
__global__ void global_step(array_ops::Array &v0, array_ops::Array &v1, array_ops::Array& v2,
                            array_ops::Array& u0, array_ops::Array &u1, array_ops::Array &u2,
                            array_ops::Float s_nu, array_ops::Float s_dx, array_ops::Float s_dt, _PDEProgram pde) {
    v0 = u0;
    v1 = u1;
    v2 = u2;
    pde.step(v0,v1,v2,u0,u1,u2,s_nu,s_dx,s_dt);
    printf("%f %f %f\n", u0[0], u1[0], u2[0]);
}

int main() {

  typedef array_ops ArrayOps;

  typedef array_ops::Array Array;
  typedef array_ops::Index Index;
  typedef array_ops::Axis Axis;
  typedef array_ops::Float Float;
  typedef array_ops::Nat Nat;
  typedef array_ops::Offset Offset;
  
  examples::pde::mg_src::pde_cpp::PDEProgram pde = examples::pde::mg_src::pde_cpp::PDEProgram();

    size_t side = SIDE; //256;
    size_t array_size = side*side*side;
    size_t steps = 10;
    //Shape shape = Shape(std::vector<size_t>({ side, side, side }));
    Array u0, u1, u2;
    
    dumpsine(u0);
    dumpsine(u1);
    dumpsine(u2);
    
    Array v0 = Array();
    Array v1 = Array();
    Array v2 = Array();
    
    memcpy(v0.content, u0.content, SIDE*SIDE*SIDE*sizeof(Float));
    memcpy(v1.content, u1.content, SIDE*SIDE*SIDE*sizeof(Float));
    memcpy(v2.content, u2.content, SIDE*SIDE*SIDE*sizeof(Float));

    Float *v0_host_content, *v1_host_content, *v2_host_content, *u0_host_content, *u1_host_content, *u2_host_content;
    Float *v0_dev_content, *v1_dev_content, *v2_dev_content, *u0_dev_content, *u1_dev_content, *u2_dev_content;

      v0_host_content = v0.content;
      v1_host_content = v1.content;
      v2_host_content = v2.content;
      u0_host_content = u0.content;
      u1_host_content = u1.content;
      u2_host_content = u2.content;

      hipMalloc((void**)&v0_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
      hipMalloc((void**)&v1_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
      hipMalloc((void**)&v2_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
      hipMalloc((void**)&u0_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
      hipMalloc((void**)&u1_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
      hipMalloc((void**)&u2_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);

      Array  *v0_dev, *v1_dev, *v2_dev, *u0_dev, *u1_dev, *u2_dev;
	
      hipMalloc((void**)&v0_dev, sizeof(*v0_dev));
      hipMalloc((void**)&v1_dev, sizeof(*v1_dev));
      hipMalloc((void**)&v2_dev, sizeof(*v2_dev));
      hipMalloc((void**)&u0_dev, sizeof(*u0_dev));
      hipMalloc((void**)&u1_dev, sizeof(*u1_dev));
      hipMalloc((void**)&u2_dev, sizeof(*u2_dev));

      hipMemcpy(v0_dev_content, v0_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(v1_dev_content, v1_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(v2_dev_content, v2_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(u0_dev_content, u0_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(u1_dev_content, u1_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(u2_dev_content, u2_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);

      hipMemcpy(&(v0_dev->content), &v0_dev_content, sizeof(u0_dev->content), hipMemcpyHostToDevice);
      hipMemcpy(&(v1_dev->content), &v1_dev_content, sizeof(u0_dev->content), hipMemcpyHostToDevice);
      hipMemcpy(&(v2_dev->content), &v2_dev_content, sizeof(u0_dev->content), hipMemcpyHostToDevice);
      hipMemcpy(&(u0_dev->content), &u0_dev_content, sizeof(u0_dev->content), hipMemcpyHostToDevice);
      hipMemcpy(&(u1_dev->content), &u1_dev_content, sizeof(u1_dev->content), hipMemcpyHostToDevice);
      hipMemcpy(&(u2_dev->content), &u2_dev_content, sizeof(u2_dev->content), hipMemcpyHostToDevice);

      //std::cout << v0_dev << std::endl;
      for (auto i = 0; i< steps; i++) {
        global_step<<<1,1>>>(*v0_dev,*v1_dev,*v2_dev,*u0_dev,*u1_dev,*u2_dev,s_nu,s_dx,s_dt, pde);

        hipDeviceSynchronize();
        hipDeviceReset();
      }
    


        
      hipMemcpy(v0_dev_content, v0_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(v1_dev_content, v1_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(v2_dev_content, v2_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);

      

      Array u0_res = Array();
      memcpy(u0_res.content, u0_host_content, sizeof(*u0_host_content) * SIDE * SIDE * SIDE);
      
      Array u1_res = Array();
      memcpy(u1_res.content, u1_host_content, sizeof(*u1_host_content) * SIDE * SIDE * SIDE);
      
      Array u2_res = Array();
      memcpy(u2_res.content, u2_host_content, sizeof(*u2_host_content) * SIDE * SIDE * SIDE);

      hipFree(v0_dev_content);
      hipFree(v1_dev_content);
      hipFree(v2_dev_content);
      hipFree(u0_dev_content);
      hipFree(u1_dev_content);
      hipFree(u2_dev_content);
      
      hipFree(v0_dev);
      hipFree(v1_dev);
      hipFree(v2_dev);
      hipFree(u0_dev);
      hipFree(u1_dev);
      hipFree(u2_dev);

      hipDeviceReset();
      exit(0);
}

