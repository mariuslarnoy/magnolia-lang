#include <vector>
#include <stdio.h>
#include <iostream>
#include "gen/examples/pde/mg-src/pde-cpp.cuh"
#include "base.cuh"

//typedef array_ops::Shape Shape;

static const double s_dt = 0.00082212448155679772495;
static const double s_nu = 1.0;
static const double s_dx = 1.0;

/*
template<>
Array forall_ops<Array,Axis,Float,Index,Nat,Offset,PDEProgram::_snippet_ix>::forall_ix_snippet_cuda(const Array &u, const Array &v,
const Array &u0, const Array &u1, const Array &u2, const Float &c0,
const Float &c1, const Float &c2, const Float &c3, const Float &c4) {

    
  }
*/

int main() {

  typedef array_ops ArrayOps;

  typedef array_ops::Array Array;
  typedef array_ops::Index Index;
  typedef array_ops::Axis Axis;
  typedef array_ops::Float Float;
  typedef array_ops::Nat Nat;
  typedef array_ops::Offset Offset;
  
  examples::pde::mg_src::pde_cpp::PDEProgram pde = examples::pde::mg_src::pde_cpp::PDEProgram();

    size_t side = SIDE; //256;
    size_t array_size = side*side*side;
    size_t steps = 50;
    //Shape shape = Shape(std::vector<size_t>({ side, side, side }));
    Array u0, u1, u2;
    
    dumpsine(u0);
    dumpsine(u1);
    dumpsine(u2);
    
    Float  *u0_host_content, *u1_host_content, *u2_host_content;

      Float *u0_dev_content, *u1_dev_content, *u2_dev_content;


      u0_host_content = u0.content;
      u1_host_content = u1.content;
      u2_host_content = u2.content;

      hipMalloc((void**)&u0_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
      hipMalloc((void**)&u1_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
      hipMalloc((void**)&u2_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);

      Array  *u0_dev, *u1_dev, *u2_dev;

      hipMalloc((void**)&u0_dev, sizeof(*u0_dev));
      hipMalloc((void**)&u1_dev, sizeof(*u1_dev));
      hipMalloc((void**)&u2_dev, sizeof(*u2_dev));

      hipMemcpy(u0_dev_content, u0_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(u1_dev_content, u1_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(u2_dev_content, u2_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);

      hipMemcpy(&(u0_dev->content), &u0_dev_content, sizeof(u0_dev->content), hipMemcpyHostToDevice);
      hipMemcpy(&(u1_dev->content), &u1_dev_content, sizeof(u1_dev->content), hipMemcpyHostToDevice);
      hipMemcpy(&(u2_dev->content), &u2_dev_content, sizeof(u2_dev->content), hipMemcpyHostToDevice);
      
      for (auto i = 0; i < steps; ++i) {
      pde.step(*u0_dev,*u1_dev,*u2_dev,s_nu,s_dx,s_dt);
      }

      hipMemcpy(u0_host_content, u0_dev_content, sizeof(*u0_host_content), hipMemcpyDeviceToHost);
      hipMemcpy(u1_host_content, u1_dev_content, sizeof(*u1_host_content), hipMemcpyDeviceToHost);
      hipMemcpy(u2_host_content, u2_dev_content, sizeof(*u2_host_content), hipMemcpyDeviceToHost);

      Array u0_res = Array();
      memcpy(u0_res.content, u0_host_content, sizeof(*u0_host_content) * SIDE * SIDE * SIDE);
      
      Array u1_res = Array();
      memcpy(u1_res.content, u1_host_content, sizeof(*u1_host_content) * SIDE * SIDE * SIDE);
      
      Array u2_res = Array();
      memcpy(u2_res.content, u2_host_content, sizeof(*u2_host_content) * SIDE * SIDE * SIDE);

      hipFree(u0_dev_content);
      hipFree(u1_dev_content);
      hipFree(u2_dev_content);

      hipFree(u0_dev);
      hipFree(u1_dev);
      hipFree(u2_dev);

      hipDeviceReset();
      exit(0);


     //   pde.step(u0,u1,u2,s_nu,s_dx,s_dt);
        
     //   std::cout << u0[0] << " "
     //             << u1[0] << " "
     //             << u2[0] << std::endl;
        
    }
    //std::cout << "after loop" << std::endl;

    //for (auto i = 0; i < SIDE*SIDE*SIDE; ++i) {
    //  std::cout << u0[i] << std::endl;
    //}
    //return 0;
    /*
    
    It's like: step { snippet { parallelize ix computations here } }
    */
