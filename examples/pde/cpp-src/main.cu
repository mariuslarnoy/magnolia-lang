#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include <iostream>
//#include "gen/examples/pde/mg-src/pde-cpp.cuh"
#include "base.cuh"

static const double s_dt = 0.00082212448155679772495;
static const double s_nu = 1.0;
static const double s_dx = 1.0;
static const size_t steps = 10;

  typedef array_ops ArrayOps;

  typedef array_ops::Array Array;
  typedef array_ops::Index Index;
  typedef array_ops::Axis Axis;
  typedef array_ops::Float Float;
  typedef array_ops::Nat Nat;
  typedef array_ops::Offset Offset;

  typedef forall_ops<Array, Axis, Float, Index, Nat, Offset> ForallOps;

  __global__ void global_step(Array *u0, Array *u1, Array *u2,
                            Float s_nu, Float s_dx, Float s_dt) {
	if(threadIdx.x == 0) {
          ForallOps forall_ops;
	  forall_ops.step(*u0,*u1,*u2,s_nu,s_dx,s_dt);
	}
}

__global__ void ix_snippet_global(array_ops::Array res, const array_ops::Array u, const array_ops::Array v, const array_ops::Array u0, const array_ops::Array u1, const array_ops::Array u2,
  const array_ops::Float c0,
    const array_ops::Float c1,
      const array_ops::Float c2,
        const array_ops::Float c3,
          const array_ops::Float c4) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < SIDE*SIDE*SIDE) {
    res[i] = snippet_cuda(u, v, u0, u1, u2, c0, c1, c2, c3, c4, i);
  }
}

void allocateDeviceMemory(Float* &u0_host_content, 
                          Float* &u1_host_content,    
                          Float* &u2_host_content,
		                      Float* &u0_dev_content, 
                          Float* &u1_dev_content, Float* &u2_dev_content,
		          Array* &u0_dev, Array* &u1_dev, Array* &u2_dev) {

      hipMalloc((void**)&u0_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
      hipMalloc((void**)&u1_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
      hipMalloc((void**)&u2_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);

      hipMalloc((void**)&u0_dev, sizeof(*u0_dev));
      hipMalloc((void**)&u1_dev, sizeof(*u1_dev));
      hipMalloc((void**)&u2_dev, sizeof(*u2_dev));

}

void copyDeviceMemory(Float* &u0_host_content, 
                      Float* &u1_host_content,    
                      Float* &u2_host_content,
                          Float* &u0_dev_content, 
                      Float* &u1_dev_content, Float* &u2_dev_content,
              Array* &u0_dev, Array* &u1_dev, Array* &u2_dev) {

      hipMemcpy(u0_dev_content, u0_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(u1_dev_content, u1_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(u2_dev_content, u2_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);

      // Binding pointers with _dev
      hipMemcpy(&(u0_dev->content), &u0_dev_content, sizeof(u0_dev->content), hipMemcpyHostToDevice);
      hipMemcpy(&(u1_dev->content), &u1_dev_content, sizeof(u1_dev->content), hipMemcpyHostToDevice);
      hipMemcpy(&(u2_dev->content), &u2_dev_content, sizeof(u2_dev->content), hipMemcpyHostToDevice);

}

int main(void) {
    
    array_ops ArrayOps;

    Float c0 = ArrayOps.div(ArrayOps.div(1.0, 2.0), s_dx);
    Float c1 = ArrayOps.div(ArrayOps.div(1.0, s_dx), s_dx);
    Float c2 = ArrayOps.div(ArrayOps.div(2.0, s_dx), s_dx);
    Float c3 = s_nu;
    Float c4 = ArrayOps.div(s_dt, 2.0);

    size_t side = SIDE; //256;
    size_t array_size = side*side*side;
    std::cout << "Dims: " << side << "*" << side << "*" << side << ", steps: " << steps << std::endl;
    
    size_t mf, ma;
    hipMemGetInfo(&mf,&ma);
    std::cout << "free: " << mf << " total: " << ma << std::endl;
  
    Array u0, u1, u2;
    
    dumpsine(u0);
    dumpsine(u1);
    dumpsine(u2);
    
    // Allocate host data 
    Float *u0_host_content, *u1_host_content, *u2_host_content;
    u0_host_content = u0.content;
    u1_host_content = u1.content;
    u2_host_content = u2.content;
    
    // Allocate device data
    Float *u0_dev_content, *u1_dev_content, *u2_dev_content;

    // Allocate device side helper structs
    Array *u0_dev, *u1_dev, *u2_dev;

    for (auto i = 0; i < steps; ++i) {

      allocateDeviceMemory(u0_host_content, u1_host_content, u2_host_content,
		           u0_dev_content, u1_dev_content, u2_dev_content,
			   u0_dev, u1_dev, u2_dev);

    } 

    
    hipMemGetInfo(&mf,&ma);
    std::cout << "free: " << mf << " total: " << ma << std::endl;

    hipDeviceReset();
    exit(0);
}

