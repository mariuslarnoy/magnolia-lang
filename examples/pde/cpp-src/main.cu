#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include <iostream>
#include "gen/examples/pde/mg-src/pde-cpp.cuh"
#include "base.cuh"

//typedef array_ops::Shape Shape;

static const double s_dt = 0.00082212448155679772495;
static const double s_nu = 1.0;
static const double s_dx = 1.0;
static const size_t steps = 10;

template<class _PDEProgram>
__global__ void global_step(array_ops::Array *u0, array_ops::Array *u1, array_ops::Array *u2,
                            const array_ops::Float s_nu, const array_ops::Float s_dx, const array_ops::Float s_dt, _PDEProgram pde) {
	if(threadIdx.x == 0) {
	  printf("%f %f %f \n", u0[0], u1[0], u2[0]);
	  //pde.step(*u0,*u1,*u2,s_nu,s_dx,s_dt);
	  
	}
}

int main(void) {
  
  typedef array_ops ArrayOps;

  typedef array_ops::Array Array;
  typedef array_ops::Index Index;
  typedef array_ops::Axis Axis;
  typedef array_ops::Float Float;
  typedef array_ops::Nat Nat;
  typedef array_ops::Offset Offset;
  
  examples::pde::mg_src::pde_cpp::PDEProgram pde = examples::pde::mg_src::pde_cpp::PDEProgram();

    size_t side = SIDE; //256;
    size_t array_size = side*side*side;
    std::cout << "Dims: " << side << "*" << side << "*" << side << ", steps: " << steps << std::endl;
    
    size_t mf, ma;
    hipMemGetInfo(&mf,&ma);
    std::cout << "free: " << mf << " total: " << ma << std::endl;
    
    Array u0, u1, u2;
    
    dumpsine(u0);
    dumpsine(u1);
    dumpsine(u2);
    
    for (auto i = 0; i < steps; ++i) {
      
      // Allocate host data
      Float *u0_host_content, *u1_host_content, *u2_host_content;
      
      u0_host_content = u0.content;
      u1_host_content = u1.content;
      u2_host_content = u2.content;

      // Allocate device data
      Float *u0_dev_content, *u1_dev_content, *u2_dev_content;
      
      hipMalloc((void**)&u0_dev_content, sizeof(Float) * array_size);
      hipMalloc((void**)&u1_dev_content, sizeof(Float) * array_size);
      hipMalloc((void**)&u2_dev_content, sizeof(Float) * array_size);
      

      // Allocate device side helper structs
      Array *u0_dev, *u1_dev, *u2_dev;
        
      hipMalloc((void**)&u0_dev, sizeof(*u0_dev));
      hipMalloc((void**)&u1_dev, sizeof(*u1_dev));
      hipMalloc((void**)&u2_dev, sizeof(*u2_dev));
    
      // Copy data from host to device
      hipMemcpy(u0_dev_content, u0_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(u1_dev_content, u1_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      hipMemcpy(u2_dev_content, u2_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);

      // Binding pointers with _dev
      hipMemcpy(&(u0_dev->content), &u0_dev_content, sizeof(u0_dev->content), hipMemcpyHostToDevice);
      hipMemcpy(&(u1_dev->content), &u1_dev_content, sizeof(u1_dev->content), hipMemcpyHostToDevice);
      hipMemcpy(&(u2_dev->content), &u2_dev_content, sizeof(u2_dev->content), hipMemcpyHostToDevice);

      // Launch parent kernel
      global_step<<<1,1>>>(u0_dev,u1_dev,u2_dev,s_nu,s_dx,s_dt, pde);
      hipDeviceSynchronize();

      // Copy u0, u1, u2 back to CPU
      hipMemcpy(u0_host_content, u0_dev_content, sizeof(*u0_host_content) * array_size, hipMemcpyDeviceToHost);
      hipMemcpy(u1_host_content, u1_dev_content, sizeof(*u1_host_content) * array_size, hipMemcpyDeviceToHost);      
      hipMemcpy(u2_host_content, u2_dev_content, sizeof(*u2_host_content) * array_size, hipMemcpyDeviceToHost);

      // Reset device memory
      hipDeviceReset();
      hipMemGetInfo(&mf,&ma);
      std::cout << "free: " << mf << " total: " << ma << std::endl;

    }
    
    /* 
    Array v0, v1, v2;

    zeros(v0);
    zeros(v1);
    zeros(v2);
    
    memcpy(v0.content, u0.content, SIDE*SIDE*SIDE*sizeof(Float));
    memcpy(v1.content, u1.content, SIDE*SIDE*SIDE*sizeof(Float));
    memcpy(v2.content, u2.content, SIDE*SIDE*SIDE*sizeof(Float));
    
    // Allocate host data 
    Float *v0_host_content, *v1_host_content, *v2_host_content, *u0_host_content, *u1_host_content, *u2_host_content;
      	
    v0_host_content = v0.content;
    v1_host_content = v1.content;
    v2_host_content = v2.content;
    u0_host_content = u0.content;
    u1_host_content = u1.content;
    u2_host_content = u2.content;
    
    // Allocate device data
    Float *v0_dev_content, *v1_dev_content, *v2_dev_content, *u0_dev_content, *u1_dev_content, *u2_dev_content;

    hipMalloc((void**)&v0_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
    hipMalloc((void**)&v1_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
    hipMalloc((void**)&v2_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
    hipMalloc((void**)&u0_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
    hipMalloc((void**)&u1_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
    hipMalloc((void**)&u2_dev_content, sizeof(Float) * SIDE * SIDE * SIDE);
      
    // Allocate device side helper structs
    Array  *v0_dev, *v1_dev, *v2_dev, *u0_dev, *u1_dev, *u2_dev;
	
    hipMalloc((void**)&v0_dev, sizeof(*v0_dev));
    hipMalloc((void**)&v1_dev, sizeof(*v1_dev));
    hipMalloc((void**)&v2_dev, sizeof(*v2_dev));
    hipMalloc((void**)&u0_dev, sizeof(*u0_dev));
    hipMalloc((void**)&u1_dev, sizeof(*u1_dev));
    hipMalloc((void**)&u2_dev, sizeof(*u2_dev));

    // Copy data from host to device
    hipMemcpy(v0_dev_content, v0_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
    hipMemcpy(v1_dev_content, v1_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
    hipMemcpy(v2_dev_content, v2_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
    hipMemcpy(u0_dev_content, u0_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
    hipMemcpy(u1_dev_content, u1_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
    hipMemcpy(u2_dev_content, u2_host_content, sizeof(Float) * SIDE * SIDE * SIDE, hipMemcpyHostToDevice);
      
    // Binding pointers with _dev
    hipMemcpy(&(v0_dev->content), &v0_dev_content, sizeof(u0_dev->content), hipMemcpyHostToDevice);
    hipMemcpy(&(v1_dev->content), &v1_dev_content, sizeof(u0_dev->content), hipMemcpyHostToDevice);
    hipMemcpy(&(v2_dev->content), &v2_dev_content, sizeof(u0_dev->content), hipMemcpyHostToDevice);
    hipMemcpy(&(u0_dev->content), &u0_dev_content, sizeof(u0_dev->content), hipMemcpyHostToDevice);
    hipMemcpy(&(u1_dev->content), &u1_dev_content, sizeof(u1_dev->content), hipMemcpyHostToDevice);
    hipMemcpy(&(u2_dev->content), &u2_dev_content, sizeof(u2_dev->content), hipMemcpyHostToDevice);
      
    // Launch parent kernel
    global_step<<<1,1>>>(*v0_dev,*v1_dev,*v2_dev,*u0_dev,*u1_dev,*u2_dev,s_nu,s_dx,s_dt, pde);
    hipDeviceSynchronize();


    hipMemGetInfo(&mf,&ma);
    std::cout << "free: " << mf << " total: " << ma << std::endl;
*/
    hipDeviceReset();
    exit(0);
}

